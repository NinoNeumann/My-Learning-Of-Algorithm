//
// Created by Nino Neumann on 2023/3/1.
//
// 我自己的思路：给一个结构体 然后排序然后想某一题 合并区间
// 并查集


#include <hip/hip_runtime.h>
#include<iostream>
#include<algorithm>
using namespace std;

const int N = 2e5;
struct range{
    int l,r;
    bool operator< (const range &t) const{
        return l<t.l;
    }
};
range ranges[N];
//vector<range> res;

int main(){
    int n;
    int res = 1;

    cin>>n;
    for(int i = 0;i<n;++i) scanf("%d%d",&ranges[i].l,&ranges[i].r);
    sort(ranges,ranges+n);
//    int ed = ranges[0].r,st = ranges[0].l;
    int maxr = ranges[0].r;
    for(int i = 1;i<n;++i){
        // 遇到问题：依次枚举 类似贪心的做法不行 缺少一些状态来告知我当前区间的合并状况
//        if(ranges[i].l>ed){
//            res++;
//            st = ranges[i].l;
//            ed = ranges[i].r;
//        }else if(ranges[i].l<ed){
//            ed = ranges[i].r;
//        }
        if (ranges[i].l<=maxr) maxr = max(maxr,ranges[i].r);
        else{
            res++;
            maxr = ranges[i].r;
        }

    }
    cout<<res<<endl;
    return 0;
}


// 下面的代码为啥不能AC
//#include<iostream>
//#include<algorithm>
//using namespace std;
//
//const int N = 2e5;
//struct range{
//    int l,r;
//    bool operator< (const range &t) const{
//        return l<t.l;
//    }
//};
//range ranges[N];
////vector<range> res;
//
//int main(){
//    int n;
//    int res = 0;
//
//    cin>>n;
//    for(int i = 0;i<n;++i) scanf("%d%d",&ranges[i].l,&ranges[i].r);
//    sort(ranges,ranges+n);
//    int ed = ranges[0].r,st = ranges[0].l;
//    for(int i = 1;i<n;++i){
//        // 遇到问题：依次枚举 类似贪心的做法不行 缺少一些状态来告知我当前区间的合并状况
//        if(ranges[i].l>ed){
//            res++;
//            st = ranges[i].l;
//            ed = ranges[i].r;
//        }else if(ranges[i].l<ed){
//            ed = ranges[i].r;
//        }
//
//    }
//    cout<<res<<endl;
//    return 0;
//}