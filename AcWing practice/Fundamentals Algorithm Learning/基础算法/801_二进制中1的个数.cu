//
// Created by Nino Neumann on 2023/3/1.
//


#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
const int N = 1e5+10;
int f[N];

int get_bits(int x){
    // 返回x二进制中1的个数
    int res = 0;
    while(x){
        if(x&1)res++;
        x >>= 1;
    }
    return res;
}

int main(){
    int n;
    cin>>n;
    for(int i = 0;i<n;++i){
        int t;
        cin>> t;
        f[i] = get_bits(t);
    }
    for(int i = 0;i<n;++i)cout<<f[i]<<" ";


    return 0;
}
